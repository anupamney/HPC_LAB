
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;
__managed__ int ans=1;
__global__ void Factorial(int *gpu_num,long int *gpu_res)
{
//	int id = threadIdx.x;
  *gpu_res=1;
  for(int i=1;i<=*gpu_num;i++)
  {
    *gpu_res = *gpu_res * i;      
  }
}

int main()
{
  int Number;
  int *dev_number;
  long int *res, result;
  system("clear");
  printf("\n\t Enter the number : ");
  scanf("%d",&Number);
  hipMalloc((void**)&dev_number,sizeof(int));
  hipMalloc((void**)&res,sizeof(long int));
  

  hipMemcpy(dev_number,&Number,sizeof(int),hipMemcpyHostToDevice);
 

  Factorial<<<1,100>>>(dev_number,res);
	
  hipMemcpy(&result,res,sizeof(long int),hipMemcpyDeviceToHost);


  printf("\n\t Factorial of number %d is %ld \n",Number,result);  
  hipFree(dev_number);
  return 0; 
}

